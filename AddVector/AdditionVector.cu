#include <iostream>
#include <math.h>
#include <time.h>
#include <stdexcept>
#include "hip/hip_runtime.h"


/* 
- Le Kernel est la fonction qui s'exécuter sur le GPU.
- les variables hôte (CPU) ont le suffixe _host de celles de périphérique (GPU) ont  _perif.
*/
// kernel
__global__ void AdditionVector(const float* A, const float* B, float* C,int N){
// calculer l'id unique pour caque thread , il doit savoir qui il est 
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  //l'addition que doit établir chaque thread 
  if (tid < N) C[tid] = A[tid] + B[tid];
}



//Fonction pour remplir les vecteur
void generateRandomVector(float *v, int dim, int lowVl, int upVal) {
    int j;
   
        for (j = 0; j < dim; ++j) {

          //v[j]=5;
         v[j] = (rand() % (upVal - lowVl + 1)) + lowVl;
        
    }

}

/***Déclaration***/
int N = 131072 ;//doit etre une puissance de 2
int NombreThread = 128 ;//doit etre une puissance de 2
int NombreBloc = N/NombreThread ;
size_t size = N * sizeof(float);





int main() {

//Déclarer les vecteurs d'entrée dans la mémoire du periphérique (GPU)
float *A_perif,*B_perif,*C_perif;
// allouer des vecteurs du periphérique dans la mémoire (GPU)
  hipMallocManaged(&A_perif, size);
  hipMallocManaged(&C_perif, size);
  hipMallocManaged(&B_perif, size);
//Déclarer et allouer les vecteurs d'entrée dans la mémoire de l'hôte (CPU)
 float* A_host = (float*)malloc(size);
 float* B_host = (float*)malloc(size);
 float* C_host = (float*)malloc(size);


//Remplir les vecteur Host
 generateRandomVector(A_host, N , 1, 99);	
 generateRandomVector(B_host, N , 1, 99);	

// Ici il faut copier les donnée des vecteur de la mémoire de la CPU à la mémoire GPU avec "cudaMemcpyHostToDevice"
  hipMemcpy(A_perif, A_host, size, hipMemcpyHostToDevice);
  hipMemcpy(B_perif, B_host, size, hipMemcpyHostToDevice);

 //Faire l'addition dans le GPU avec la fonction definit comme kernel



 //Appler kernel
AdditionVector<<<NombreBloc,NombreThread>>>(A_perif,B_perif,C_perif, N);




// copier le résultat obtenu en  C_perif ver C_host acev "cudaMemcpyDeviceToHost"
  hipMemcpy(C_host, C_perif, size, hipMemcpyDeviceToHost);

for (int i=0; i<N; i++) { 
printf("vector C : %f\n", C_host[i]);
    }

printf("NombreBloc : %d\n", NombreBloc);

  // libérer l'éspace allouer en GPU et CPU
  hipFree(A_perif);
  hipFree(B_perif);
  hipFree(C_perif);






  free(A_host);
  free(B_host);
  free(C_host);


return hipDeviceSynchronize();

}
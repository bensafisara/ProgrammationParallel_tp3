#include <iostream>
#include <math.h>
#include <time.h>
#include <stdexcept>
#include "hip/hip_runtime.h"

/***Déclaration***/
int N=2048;
int NombreThread = 64;//doit etre une puissance de 2
size_t size = N* N * sizeof(int);


/* 
- Le Kernel est la fonction qui s'exécuter sur le GPU.
- les variables hôte (CPU) ont le suffixe _host de celles de périphérique (GPU) ont  _perif.
*/

// kernel
__global__ void AdditionMatrice(int *A, int *B, int *C,int N){

   //ICI on a une matrice de deux dimension chaque bloc aura deux ID donc, le tid1 et tid2
  //il faudra calculé le id pour chacque bloc
     int tid1 = blockIdx.x * blockDim.x + threadIdx.x;
     int tid2 = blockIdx.y * blockDim.y + threadIdx.y;
    
    //Ce calcule est une façon pour accéder à chaque bloc 
    int id=tid2*N+tid1;

    //l'addition que doit établir chaque thread 
    if (tid1 <N &&  tid2< N)
      C[id] = A[id]+ B[id];

}

//Fonction pour remplir les vecteur
 void generateRandomMatrix(int *M, int dim, int lowVal, int upVal) {
      for (int i = 0; i < dim; ++i) {
          for (int j = 0; j < dim; ++j) {
           M[i*dim+j] = (rand() % (upVal - lowVal + 1)) + lowVal;
      }}}


int main() {

//Déclarer les vecteurs d'entrée dans la mémoire du periphérique (GPU)
int *A_perif,*B_perif,*C_perif;


//Déclarer et allouer les matrice d'entrée dans la mémoire de l'hôte (CPU)
    int *A_host=(int*)malloc(sizeof(int) * (N*N));
    int *B_host=(int*)malloc(sizeof(int) * (N*N));
    int *C_host=(int*)malloc(sizeof(int) * (N*N));


//Allouer des vecteurs du periphérique dans la mémoire (GPU)
  hipMalloc(&A_perif, (N*N )*sizeof(int));
  hipMalloc(&C_perif, (N*N )*sizeof(int));
  hipMalloc(&B_perif, (N*N )*sizeof(int));


//Remplir les vecteur Host
 generateRandomMatrix(A_host, N , 1,50); 
 generateRandomMatrix(B_host, N , 1,20);

//Ici il faut copier les donnée des Matrice de la mémoire de la CPU à la mémoire GPU avec "cudaMemcpyHostToDevice"
  hipMemcpy(A_perif, A_host, size, hipMemcpyHostToDevice);
  hipMemcpy(B_perif, B_host, size, hipMemcpyHostToDevice);
  hipMemcpy(C_perif, C_host, size, hipMemcpyHostToDevice);

 //Faire l'addition dans le GPU avec la fonction definit comme kernel


//Définir les paramètre du kernel 
dim3 bD(NombreThread,NombreThread);
int NombreBloc = (N +  NombreThread -1) / NombreThread;
dim3 gD( NombreBloc,NombreBloc);

//Appler kernel
AdditionMatrice<<< gD ,bD>>>(A_perif,B_perif,C_perif, N);


//copier le résultat obtenu en  C_perif ver C_host avec "cudaMemcpyDeviceToHost"
  hipMemcpy(C_host, C_perif, size, hipMemcpyDeviceToHost);

  
//Afficher le résultat de l'addition
for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("Matrice C : %d",C_host[i*N+j]);}
       printf("\n");}
//Afficher Nombre Bloc
printf(" \n bloc ::: %d \n", NombreBloc);

// libérer l'éspace allouer en GPU et CPU
  hipFree(A_perif);
  hipFree(B_perif);
  hipFree(C_perif);

  free(A_host);
  free(B_host);
  free(C_host);


return hipDeviceSynchronize();

}
#include <iostream>
#include <math.h>
#include <time.h>
#include <stdexcept>
#include "hip/hip_runtime.h"

/***Déclarations***/
int N=4096;//16777216 elements
int NombreThread = 8;//puissance de 2
size_t size = N* N * sizeof(int);


/* 
- Le Kernel est la fonction qui s'exécuter sur le GPU.
- les variables hôte (CPU) ont le suffixe _host de celles de périphérique (GPU) ont  _perif.
*/

// kernel
__global__ void AdditionMatrice(int *A, int *B, int *C,int N){

  //ICI on a une matrice de deux dimension chaque bloc aura deux ID donc, le tid1 et tid2
  //il faudra calculé le id pour chacque bloc
     int tid1 = blockIdx.x * blockDim.x + threadIdx.x;
     int tid2 = blockIdx.y * blockDim.y + threadIdx.y;
    
    C[tid2*N+tid1]=0;

  //Ce calcule est une façon pour accéder à chaque bloc 
  //l'addition que doit établir chaque thread 
    for (int i = 0; i < N; ++i){
       C[tid2*N+tid1] += A[tid2*N+i]* B[N*i+tid1];
    }
  }


//Fonction pour remplir les vecteurs
  void generateRandomMatrix(int *M, int dim, int lowVal, int upVal) {
      for (int i = 0; i < dim; ++i) {
          for (int j = 0; j < dim; ++j) {
           M[i*dim+j] = (rand() % (upVal - lowVal + 1)) + lowVal;
      }}}

int main() {

//Déclarer les vecteurs d'entrée dans la mémoire du periphérique (GPU)
    int *A_perif,*B_perif,*C_perif;


//Déclarer et allouer les matrice d'entrée dans la mémoire de l'hôte (CPU)
    int *A_host=(int*)malloc(sizeof(int) * (N*N));
    int *B_host=(int*)malloc(sizeof(int) * (N*N));
    int *C_host=(int*)malloc(sizeof(int) * (N*N));

//Allouer les Matrices du periphérique dans la mémoire (GPU)

    hipMalloc(&A_perif, (N*N )*sizeof(int));
    hipMalloc(&C_perif, (N*N )*sizeof(int));
    hipMalloc(&B_perif, (N*N )*sizeof(int));

//Remplir les Matrices Host
    generateRandomMatrix(A_host, N , 1,99); 
    generateRandomMatrix(B_host, N , 1,99);

// Ici il faut copier les donnée des Matrices de la mémoire de la CPU à la mémoire GPU avec "cudaMemcpyHostToDevice"
    hipMemcpy(A_perif, A_host, size, hipMemcpyHostToDevice);
    hipMemcpy(B_perif, B_host, size, hipMemcpyHostToDevice);
    hipMemcpy(C_perif, C_host, size, hipMemcpyHostToDevice);

//Faire l'addition dans le GPU avec la fonction definit comme kernel
   
       //bD correspond au nombre de threads par block.
      //gD correspond au nombre de block de threads dans une grille
     //Définir gD bD
      dim3 bD(NombreThread,NombreThread);

      int NombreBloc = (N +  NombreThread -1) / NombreThread;
      printf(" bloc ::: %d \n", NombreBloc);
      dim3 gD( NombreBloc,NombreBloc);

     //Appler kernel
      AdditionMatrice<<< gD ,bD>>>(A_perif,B_perif,C_perif, N);


/*copier le résultat obtenu en  C_perif ver C_host avec "cudaMemcpyDeviceToHost"*/
hipMemcpy(C_host, C_perif, size, hipMemcpyDeviceToHost);
for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("\nMatrice C : %d",C_host[i*N+j]);

}
  printf("\n");
}

 printf(" \n bloc ::: %d \n", NombreBloc);
// libérer l'éspace allouer en GPU et CPU
  hipFree(A_perif);
  hipFree(B_perif);
  hipFree(C_perif);

free(A_host);
free(B_host);
free(C_host);
//Attendre que le GPU termine
hipDeviceSynchronize();
return 0;

}